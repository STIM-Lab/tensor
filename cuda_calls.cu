#include <hip/hip_runtime.h>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <tira/cuda/cudaGaussianFilter2D.cuh>


glm::mat2* cudaGaussianBlur(glm::mat2* source, unsigned int width, unsigned int height, float sigma,
                            unsigned int& out_width, unsigned int& out_height, int deviceID = 0) {

    hipSetDevice(deviceID);
    glm::mat2* dest = GaussianFilter2D<glm::mat2>(source, width, height, sigma, sigma, out_width, out_height);

    return dest;
}