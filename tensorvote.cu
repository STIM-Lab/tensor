#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "tensorvote.cuh"

#include <chrono>

#ifdef __HIPCC__
#define __device__
#endif

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        std::cout << hipGetErrorString(err) << "in" << file << "at line" << line << std::endl;
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

tira::image<glm::mat2> voteCPU(tira::image<glm::mat2> T, int sigma)
{
    int w = 6 * sigma / 2;

    int X = T.shape()[1];
    int Y = T.shape()[0];

    tira::image<glm::mat2> VT(X, Y);

    for (int x = 0; x < X; x++)
    {
        for (int y = 0; y < Y; y++)
        {
            for (int u = -w; u < w; u++)
            {
                for (int v = -w; v < w; v++)
                {
                    if (y + v >= 0 && y + v < Y && x + u >= 0 && x + u < X)
                    {
                        VoteContribution vc = Saliency(T(x, y), u, v, sigma);

                        VT(x + u, y + v) += vc.votes * vc.decay;
                    }
                }
            }
        }
    }

    return VT;
}

__global__ void voteGPU(float *data, float *VT, int sigma, int w, int width, int height)
{
    size_t y = blockDim.y * blockIdx.y + threadIdx.y;                                       // get the x and y image coordinates for the current thread
    size_t x = blockDim.x * blockIdx.x + threadIdx.x;

    
    if (y >= height || x >= width)                                                          // if not within bounds of image, return
        return;

    float vt[4] = {0, 0, 0, 0};                                                             // initialize a tensor to zeros

    for (int u = -w; u < w; u++)                                                            // for each pixel within the window
    {
        for (int v = -w; v < w; v++)
        {
            int index = ((v + y) * width + x + u);                                          // calculate a 1D index into the tensor field
            //int indexShared = ((threadIdx.y + v) * blockDim.y + threadIdx.x + u);
            if (index < width * height && index >= 0) {                                     // DAVID: This will cause wrap-around artifacts
                glm::mat2 T(
                    data[4 * index + 0],
                    data[4 * index + 1],
                    data[4 * index + 2],
                    data[4 * index + 3]);

                VoteContribution vc = Saliency(T, u, v, sigma);                             // calculate the saliency given the tensor at (u,v)
                vt[0] += vc.votes[0][0] * vc.decay;                                         // sum the tensor contribution based on the saliency
                vt[1] += vc.votes[0][1] * vc.decay;
                vt[2] += vc.votes[1][0] * vc.decay;
                vt[3] += vc.votes[1][1] * vc.decay;
            }
        }
    }

    VT[4 * (y * width + x) + 0] = vt[0];
    VT[4 * (y * width + x) + 1] = vt[1];
    VT[4 * (y * width + x) + 2] = vt[2];
    VT[4 * (y * width + x) + 3] = vt[3];
}

tira::image<glm::mat2> CPUImplementation(tira::image<glm::mat2> Tn, int sigma)
{
    std::cout << "**********CPU**********" << std::endl;

    std::chrono::high_resolution_clock::time_point start, stop;
    start = std::chrono::high_resolution_clock::now();

    tira::image<glm::mat2> T = voteCPU(Tn, sigma);

    stop = std::chrono::high_resolution_clock::now();
    std::chrono::milliseconds d;
    d = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Elapsed time: " << d.count() / 1000.0f << " seconds" << std::endl;

    return T;
}

/// <summary>
/// Calculates one iteration of tensor voting given an input tensor field Tn
/// </summary>
/// <param name="Tn">Input tensor field as a 2D image of matrices</param>
/// <param name="sigma">Standard deviation for the decay function</param>
/// <param name="w">Window size for the decay function (usually dependent on sigma)</param>
/// <returns></returns>
float *CUDAImplementation(tira::image<glm::mat2> Tn, int sigma, int w)
{
    hipDeviceProp_t props;
    HANDLE_ERROR(hipGetDeviceProperties(&props, 0));

    std::cout << "**********CUDA**********" << std::endl;

    float width = Tn.shape()[1];
    float height = Tn.shape()[0];
    int size = 4 * width * height;

    float *data = (float *)Tn.data();

    float *inArray;
    float *outArray;

    HANDLE_ERROR(hipMalloc(&inArray, size * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&outArray, size * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(inArray, data, size * sizeof(float), hipMemcpyHostToDevice));

    size_t blockDim = sqrt(props.maxThreadsPerBlock);
    dim3 threads(blockDim, blockDim);

    dim3 blocks(width / threads.x + 1, height / threads.y + 1);

    int sharedBytes = props.sharedMemPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    voteGPU<<<blocks, threads, sharedBytes>>>(inArray, outArray, sigma, w, width, height);              // call the CUDA kernel for voting

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float *gpu_out = new float[size];
    HANDLE_ERROR(hipMemcpy(gpu_out, outArray, size * sizeof(float), hipMemcpyDeviceToHost));

    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);

    std::cout << "Elapsed time: " << totalTime / 1000 << " seconds" << std::endl;

    return gpu_out;
}

